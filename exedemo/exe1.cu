#include "hip/hip_runtime.h"
#include <cstdio>
#include "exe3.h"
#include "../libdemo/demo3.h"

__global__ void AnotherKernel() {
    Demo d;
    int r = d.Test1(2, 3);
    printf("%d", r);
}

void Exe::AnotherLaunch() {
    AnotherKernel<<<2, 2>>>();
}