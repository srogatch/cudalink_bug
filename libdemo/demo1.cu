#include "hip/hip_runtime.h"
#include <cstdio>
#include "demo3.h"

__device__ int Demo::Test1(int a, int b) {
    return a+b;
}

__global__ void TestKernel() {
    printf("Demo");
}

void Demo::TestLaunch() {
    TestKernel<<<1,2>>>();
}